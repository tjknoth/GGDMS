#include "hip/hip_runtime.h"
// -*- c++ -*-

/* Copyright 2012 Jeffrey Blanchard, Erik Opavsky, and Emircan Uysaler
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <limits>
#include <math.h>
#include <ctime>
//#include "recursionKernels.cu"
#include "findk.cu"



namespace BucketMultiselectNewFindK{
  using namespace std;

#define MAX_THREADS_PER_BLOCK 1024
#define CUTOFF_POINT 200000 
#define TIMING_ON
#define MIN_SLOPE 2 ^ -1022
#define SAFE

#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)


  /// ***********************************************************
  /// ***********************************************************
  /// **** HELPER CPU FUNCTIONS
  /// ***********************************************************
  /// ***********************************************************



  /* This timing function uses CUDA event timing to process the amount of time
     required, and print out result with the given index.

     start a timer with option = 0
     stop a timer with option = 1
  */

  
  hipEvent_t start, stop;
  float time;

  inline void timing_switch(int option, int ind){
    if(option == 0) {
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start, 0);
    } else {
      hipDeviceSynchronize();
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);
      hipEventDestroy(start);
      hipEventDestroy(stop);
      printf("Time %d: %lf \n", ind, time);
    }
  }

  inline void timing(int option, int ind){
#ifdef TIMING_ON 
    timing_switch(option, ind);
#endif
  }


  /* This function initializes a vector to all zeros on the host (CPU).
   */
  template<typename T>
  void setToAllZero (T * d_vector, int length) {
    hipMemset(d_vector, 0, length * sizeof(T));
  }


  void Check_CUDA_Error(const char *message)
  {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "Error: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
    }
  }

  inline void SAFEcuda(const char *message) {
#ifdef SAFE
    Check_CUDA_Error(message);
#endif
  }


  /*
*********************************************************
*/


  /* This function finds the bin containing the kth element we are looking for (works on 
     the host). While doing the scan, it stores the sum-so-far of the number of elements in 
     the ctive buckets containing one of the k order statistics.

     markedBuckets : buckets containing the corresponding k values
     sums : sum-so-far of the number of elements in the buckets where k values fall into
  */
  inline int findKBuckets(uint * d_bucketCount, uint * h_bucketCount, int numBuckets
                          , uint * kVals, int numKs, uint * sums, uint * markedBuckets
                          , int numBlocks) {
    // consider the last row which holds the total counts
    int sumsRowIndex= numBuckets * (numBlocks-1);

    SAFEcuda("pre memcpy");

    CUDA_CALL(hipMemcpy(h_bucketCount, d_bucketCount + sumsRowIndex, 
                         sizeof(uint) * numBuckets, hipMemcpyDeviceToHost));
    SAFEcuda("memcpy");

    int kBucket = 0;
    int k;
    int sum = h_bucketCount[0];

    for(register int i = 0; i < numKs; i++) {
      k = kVals[i];
      while ((sum < k) & (kBucket < numBuckets - 1)) {
        kBucket++;
        sum += h_bucketCount[kBucket]; 
      }
      markedBuckets[i] = kBucket;

      sums[i] = sum - h_bucketCount[kBucket];

    }

    return 0;
  }

  // **********************************************************
  // ***********  sort  phase differs by type  ****************
  // ****** mgpu merge sort typically faster for doubles *****
  // **********************************************************
  template <typename T>
  void inline sort_phase (T* Input, const int length) {
    cubDeviceSort<T>(Input, length);
  }


  template <>
  void inline sort_phase<double> (double* Input, const int length) {
    mgpuDeviceSort<double>(Input, length);
  }


  /// ***********************************************************
  /// ***********************************************************
  /// **** HELPER GPU FUNCTIONS-KERNELS
  /// ***********************************************************
  /// ***********************************************************



  /* This function assigns elements to buckets based on the pivots and slopes determined 
     by a randomized sampling of the elements in the vector. At the same time, this 
     function keeps track of count.

     d_elementToBucket : bucket assignment for every array element
     d_bucketCount : number of element that falls into the indexed buckets within the block
  */
  template <typename T>
  __global__ void assignSmartBucket (T * d_vector, int length, int numBuckets
                                     , double * slopes, T * pivots, T * pivottree, int numPivots
                                     , uint* d_elementToBucket , uint* d_bucketCount, int offset) {
  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    uint bucketIndex;
    int threadIndex = threadIdx.x;  

    int numBigBuckets = numPivots - 1;
    
    //variables in shared memory for fast access
    __shared__ int sharedNumSmallBuckets;
    if (threadIndex < 1) 
      sharedNumSmallBuckets = numBuckets / numBigBuckets;

    extern __shared__ uint array[];
    double * sharedSlopes = (double *)array;
    T * sharedPivots = (T *)&sharedSlopes[numPivots];
    T * sharedPivotTree = (T *)&sharedPivots[numPivots];
    uint * sharedBuckets = (uint *)&sharedPivotTree[numPivots];

  
    //reading bucket counts into shared memory where increments will be performed
    for (int i = 0; i < (numBuckets / MAX_THREADS_PER_BLOCK); i++) 
      if (threadIndex < numBuckets) 
        sharedBuckets[i * MAX_THREADS_PER_BLOCK + threadIndex] = 0;

    if(threadIndex < numPivots) {
      *(sharedPivots + threadIndex) = *(pivots + threadIndex);
      *(sharedSlopes + threadIndex) = *(slopes + threadIndex);
      *(sharedPivotTree + threadIndex) = *(pivottree + threadIndex);
    }


    syncthreads();

    //assigning elements to buckets and incrementing the bucket counts
    if(index < length) {
      int i;

      for(i = index; i < length; i += offset) {
        T num = d_vector[i];

        int PivotIndex = 1;

        for(int j=1; j < numBigBuckets; j*=2){
          PivotIndex = (PivotIndex << 1) + (num >= sharedPivotTree[PivotIndex-1]);
        }
        PivotIndex = PivotIndex - numBigBuckets;

	int localBucket = (int) (((double)num - (double)sharedPivots[PivotIndex]) 
                                 * sharedSlopes[PivotIndex]);

        bucketIndex = (PivotIndex * sharedNumSmallBuckets) 
          + localBucket;
        if (bucketIndex == numBuckets) 
          bucketIndex= numBuckets-1;


        d_elementToBucket[i] = bucketIndex;
        atomicInc(sharedBuckets + bucketIndex, length); 

      }
    }

    syncthreads();      

    //reading bucket counts from shared memory back to global memory
    for (int i = 0; i <(numBuckets / MAX_THREADS_PER_BLOCK); i++) {
      if (threadIndex < numBuckets) {
        *(d_bucketCount + blockIdx.x * numBuckets 
          + i * MAX_THREADS_PER_BLOCK + threadIndex) = 
          *(sharedBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex);
      } // end if threadIndex < numBuckets
    } // end for

  } // end function assignSmartBuckets



  /* This function cumulatively sums the count of every block for a given bucket s.t. the
     last block index holds the total number of elements falling into that bucket all over the 
     array.
     updates d_bucketCount
  */
  __global__ void sumCounts(uint * d_bucketCount, const int numBuckets
                            , const int numBlocks) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j=1; j<numBlocks; j++) 
      d_bucketCount[index + numBuckets*j] += d_bucketCount[index + numBuckets*(j-1)];
    
  }



  /* This function reindexes the buckets counts for every block according to the 
     accumulated d_reindexCounter counter for the reduced vector.
     updates d_bucketCount
  */
  __global__ void reindexCounts(uint * d_bucketCount, const int numBuckets
                                , const int numBlocks, uint * d_reindexCounter
                                , uint * d_markedBuckets , const int numUniqueBuckets) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex<numUniqueBuckets) {
      int index = d_markedBuckets[threadIndex];
      uint add = d_reindexCounter[threadIndex];

      for(int j=0; j<numBlocks; j++) 
        d_bucketCount[index + numBuckets*j] += add;
    }
  }



  /* This function copies the elements of buckets that contain kVals into a newly allocated 
     reduced vector space.
     newArray - reduced size vector containing the essential elements
     *** This function is not used in current implementation, replaced by copyElements_tree. ***
     */
  template <typename T>
  __global__ void copyElements (T* d_vector, int length, uint* elementToBucket
                                , uint * buckets, const int numBuckets, T* newArray, uint offset
                                , uint * d_bucketCount, int numTotalBuckets) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex;
    int loop = numBuckets / MAX_THREADS_PER_BLOCK;

    extern __shared__ uint sharedBuckets[];

    for (int i = 0; i <= loop; i++) {      
      threadIndex = i * blockDim.x + threadIdx.x;
      if(threadIndex < numBuckets) {
        sharedBuckets[threadIndex] = buckets[threadIndex];
      }
    }
    
    syncthreads();

    int minBucketIndex;
    int maxBucketIndex; 
    int midBucketIndex;
    uint temp;
    int compare;

    if(idx < length) {
      for(int i=idx; i<length; i+=offset) {
        temp = elementToBucket[i];
        minBucketIndex = 0;
        maxBucketIndex = numBuckets-1;
        compare = 0;

        //thread divergence avoiding binary search over the markedBuckets to find a match quickly
        for(int j = 1; j < numBuckets; j*=2) {  
          midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
          compare = (temp > sharedBuckets[midBucketIndex]);
          minBucketIndex = compare ? midBucketIndex : minBucketIndex;
          maxBucketIndex = compare ? maxBucketIndex : midBucketIndex;
        }

        if (buckets[maxBucketIndex] == temp) 
          newArray[atomicDec(d_bucketCount + blockIdx.x * numTotalBuckets 
                             + sharedBuckets[maxBucketIndex], length)-1] = d_vector[i];
      }
    }

  }

  /* This kernel copies the elements of buckets that contain kVals into a newly allocated 
     reduced vector space.
     newArray - reduced size vector containing the essential elements.
     This kernel differs from copyElements in that it loads a binary search tree
     for the unique buckets into shared memory.  It requires more shared memory to properly
     form the tree.  For a small number (< 128) of order statistics, the tree search is not advantageous.
     The main bucketMultiselect can be altered to utilize an if (numUnique < 128) conditional, calling 
     copyElements if true and calling copyElements_tree if false.
  */



  template <typename T>
  __global__ void copyElements_tree (T* d_vector, int length, uint* elementToBucket
                                     , uint * uniqueBuckets, const int numUnique, const int numUnique_extended, T* newArray, uint offset
                                     , uint * d_bucketCount, int numTotalBuckets) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex;
    int loop = (numUnique_extended) / MAX_THREADS_PER_BLOCK;
    int mid = numUnique_extended / 2;
    int blockOffset = blockIdx.x * numTotalBuckets;

    extern __shared__ uint activeTree[];

    int treeidx, level, shift, remainder, bucketidx;
    // read from shared memory into a binary search tree
    for (int i = 0; i <= loop; i++) {      
      threadIndex = i * blockDim.x + threadIdx.x;
      if (threadIndex < numUnique_extended) {
        treeidx = threadIndex+1;
        level = (int) floorf ( log2f( (float)treeidx ) );
        shift = (1 << level);
        remainder = treeidx - shift;

        bucketidx = ((2*remainder + 1)*mid) / shift;
        if (bucketidx < numUnique) {
          activeTree[threadIndex] = uniqueBuckets[bucketidx];
        } else {
          activeTree[threadIndex] = uniqueBuckets[numUnique-1];
        } // end if (bucketidx) {} else
      } // end if (threadIndex)
    }  // end for
    
    syncthreads();


    int temp_bucket, temp_active, treeindex, active, searchdepth;

    // binary search tree through the active buckets to see
    // if the current element is in an active bucket.  
    // If not, active = 0. If so, active = 1.
    if(idx < length) {
      for(int i=idx; i<length; i+=offset) {
        temp_bucket = elementToBucket[i];
        treeindex = 1;
        active = 0;
        searchdepth = 1;
        while ( (active==0) && (searchdepth<numUnique_extended) ){
          temp_active = activeTree[treeindex - 1];
          searchdepth *= 2;
          (temp_active == temp_bucket) ? active++ : ( treeindex = (treeindex << 1) + (temp_bucket > temp_active) );
        }  // endwhile


        // if this element is in an active bucket, copy it to the new input vector
        if (active) {
          newArray[atomicDec(d_bucketCount + blockOffset + temp_active, length)-1] = d_vector[i];
        }  // end if (active)
      }  // ends for loop with offset jump
    } // ends if (idx < length)
  }  // ends copyElements_tree kernel





  /* This function speeds up the copying process the requested kVals by clustering them
     together.
  */
  template <typename T>
  __global__ void copyValuesInChunk (T * outputVector, T * inputVector, uint * kList
                                     , uint * kIndices, int kListCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int loop = kListCount / MAX_THREADS_PER_BLOCK;

    for (int i = 0; i <= loop; i++) {      
      if (idx < kListCount)
        *(outputVector + *(kIndices + idx)) = *(inputVector + *(kList + idx) - 1);
    }
  }


  /// ***********************************************************
  /// ***********************************************************
  /// **** GENERATE PIVOTS
  /// ***********************************************************
  /// ***********************************************************



  /* Hash function using Monte Carlo method
   */
  __host__ __device__
  unsigned int hash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
  }



  /* RandomNumberFunctor
   */
  struct RandomNumberFunctor :
    public thrust::unary_function<unsigned int, float> {
    unsigned int mainSeed;

    RandomNumberFunctor(unsigned int _mainSeed) :
    mainSeed(_mainSeed) {}
  
    __host__ __device__
    float operator()(unsigned int threadIdx)
    {
      unsigned int seed = hash(threadIdx) * mainSeed;

      thrust::default_random_engine rng(seed);
      rng.discard(threadIdx);
      thrust::uniform_real_distribution<float> u(0, 1);

      return u(rng);
    }
  };



  /* This function creates a random vector of 1024 elements in the range [0 1]
   */
  template <typename T>
  void createRandomVector(T * d_vec, int size) {
    timeval t1;
    uint seed;

    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
  
    thrust::device_ptr<T> d_ptr(d_vec);
    thrust::transform (thrust::counting_iterator<uint>(0), 
                       thrust::counting_iterator<uint>(size), 
                       d_ptr, RandomNumberFunctor(seed));
  }



  /* This function maps the [0 1] range to the [0 vectorSize] and 
     grabs the corresponding elements.
  */
  template <typename T>
  __global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
    *(in + blockIdx.x*blockDim.x + threadIdx.x) = 
      *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
  }

  __global__ void enlargeIndexAndGetElements (float * in, uint * out, uint * list, int size) {
    *(out + blockIdx.x * blockDim.x + threadIdx.x) = 
      (uint) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
  }



  /* This function generates Pivots from the random sampled data and calculates slopes.
 
     pivots - arrays of pivots
     slopes - array of slopes
  */
  template <typename T>
  void generatePivots (uint * pivots, uint * pivottree, double * slopes, uint * d_list, int sizeOfVector
                       , int numPivots, int sizeOfSample, int totalSmallBuckets, uint min, uint max) {
  
    float * d_randomFloats;
    uint * d_randomInts;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);
  
    d_randomInts = (uint *) d_randomFloats;

    createRandomVector (d_randomFloats, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
      , MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list, 
                                 sizeOfVector);

    pivots[0] = min;
    pivots[numPivots-1] = max;

    cubDeviceSort<T>(d_randomInts, sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are next to the min and max pivots using the random element 
    // endOffset away from the ends
    hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (uint)
                , hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1, 
                sizeof (uint), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

    for (register int i = 2; i < numPivots - 2; i++) {
      hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1, 
                  sizeof (uint), hipMemcpyDeviceToHost);
      slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
    }

    slopes[numPivots - 3] = numSmallBuckets / 
      (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets / 
      (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

    int level = numPivots - 1;
    int shift = 0;
    for (register int j=1; j < (numPivots - 1); j*=2){
      level >>= 1;
      for (register int k=0; k <= shift; k++){
        pivottree[shift + k] = pivots[(2*k+1)*level];
      }
      shift = (shift << 1) | 1;
    }

    hipFree(d_randomFloats);
  }  // end generatePivots
  
  template <typename T>
  void generatePivots (T * pivots, T * pivottree, double * slopes, T * d_list, int sizeOfVector
                       , int numPivots, int sizeOfSample, int totalSmallBuckets, T min, T max) {
    T * d_randoms;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randoms, sizeof (T) * sizeOfSample);
  
    createRandomVector (d_randoms, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
      , MAX_THREADS_PER_BLOCK>>>(d_randoms, d_list, sizeOfVector);

    pivots[0] = min;
    pivots[numPivots - 1] = max;

    cubDeviceSort<T>(d_randoms, sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are endOffset away from the min and max pivots
    hipMemcpy (pivots + 1, d_randoms + endOffset - 1, sizeof (T), 
                hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randoms + sizeOfSample - endOffset - 1, 
                sizeof (T), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / ((double)pivots[1] - (double)pivots[0]);
    
    for (register int i = 2; i < numPivots - 2; i++) {
      hipMemcpy (pivots + i, d_randoms + pivotOffset * (i - 1) + endOffset - 1, 
                  sizeof (T), hipMemcpyDeviceToHost);
      slopes[i - 1] = numSmallBuckets / ((double) pivots[i] - (double) pivots[i - 1]);
    }

    slopes[numPivots - 3] = numSmallBuckets / 
      ((double)pivots[numPivots - 2] - (double)pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets / 
      ((double)pivots[numPivots - 1] - (double)pivots[numPivots - 2]);

    // **** extra space in slopes
    slopes[numPivots - 1]=0;

    int level = numPivots - 1;
    int shift = 0;
    for (register int j=1; j < (numPivots - 1); j*=2){
      level >>= 1;
      for (register int k=0; k <= shift; k++){
        pivottree[shift + k] = pivots[(2*k+1)*level];
      }
      shift = (shift << 1) | 1;
    }
        
    hipFree(d_randoms);
  } // end generatePivots<uint>



  /// ***********************************************************
  /// ***********************************************************
  /// **** bucketMultiSelect: the main algorithm
  /// ***********************************************************
  /// ***********************************************************

  /* This function is the main process of the algorithm. It reduces the given multi-selection
     problem to a smaller problem by using bucketing ideas.
  */
  template <typename T>
  T bucketMultiSelect (T* d_vector, int length, uint * kVals, int numKs, T * output, int blocks
                       , int threads, int numBuckets, int numPivots) {    

    //CUDA_CALL(hipDeviceReset());

    /// ***********************************************************
    /// **** STEP 1: Initialization 
    /// **** STEP 1.1: Find Min and Max of the whole vector
    /// **** We don't need to go through the rest of the algorithm if it's flat
    /// ***********************************************************
    timing(0,1);



    //find max and min with thrust
    T maximum, minimum;

    thrust::device_ptr<T>dev_ptr(d_vector);
    thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result = 
      thrust::minmax_element(dev_ptr, dev_ptr + length);

    minimum = *result.first;
    maximum = *result.second;

    //if the max and the min are the same, then we are done
    if (maximum == minimum) {
      for (register int i = 0; i < numKs; i++) 
        output[i] = minimum;
      
      return 1;
    }

    /// ***********************************************************
    /// **** STEP 1: Initialization 
    /// **** STEP 1.2: Declare variables and allocate memory
    /// **** Declare Variables
    /// ***********************************************************

    //declaring variables for kernel launches
    int threadsPerBlock = threads;
    int numBlocks = blocks;
    int offset = blocks * threads;

    // variables for the randomized selection
    int sampleSize = 1024;

    // pivot variables
    double slopes[numPivots - 1];
    double * d_slopes;
    T pivots[numPivots];
    T * d_pivots;
    T pivottree[numPivots];
    T * d_pivottree;

    //Allocate memory to store bucket assignments
    size_t size = length * sizeof(uint);
    uint * d_elementToBucket;    //array showing what bucket every element is in

    CUDA_CALL(hipMalloc(&d_elementToBucket, size));

    //Allocate memory to store bucket counts
    size_t totalBucketSize = numBlocks * numBuckets * sizeof(uint);
    uint * h_bucketCount = (uint *) malloc (numBuckets * sizeof (uint));

    //array showing the number of elements in each bucket
    uint * d_bucketCount; 

    CUDA_CALL(hipMalloc(&d_bucketCount, totalBucketSize));

    // array of kth buckets
    int numUniqueBuckets;
    uint * d_kVals; 
    uint kthBuckets[numKs]; 
    uint kthBucketScanner[numKs]; 
    uint * kIndices = (uint *) malloc (numKs * sizeof (uint));
    uint * d_kIndices;
    uint uniqueBuckets[numKs];
    uint * d_uniqueBuckets; 
    uint reindexCounter[numKs];  
    uint * d_reindexCounter;    
    //    int precount;


    CUDA_CALL(hipMalloc(&d_kVals, numKs * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_kIndices, numKs * sizeof (uint)));

    for (register int i = 0; i < numKs; i++) {
      kthBucketScanner[i] = 0;
      kIndices[i] = i;
    }

    // variable to store the end result
    int newInputLength, newInputLengthAlt;
    T* newInput;
    T* newInputAlt;


    /// ***********************************************************
    /// **** STEP 1: Initialization 
    /// **** STEP 1.3: Sort the klist
    /// and keep the old index
    /// ***********************************************************

    CUDA_CALL(hipMemcpy(d_kIndices, kIndices, numKs * sizeof (uint), 
                         hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_kVals, kVals, numKs * sizeof (uint), 
                         hipMemcpyHostToDevice)); 

    // sort the given indices
    thrust::device_ptr<uint>kVals_ptr(d_kVals);
    thrust::device_ptr<uint>kIndices_ptr(d_kIndices);
    thrust::sort_by_key(kVals_ptr, kVals_ptr + numKs, kIndices_ptr);

    CUDA_CALL(hipMemcpy(kIndices, d_kIndices, numKs * sizeof (uint), 
                         hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(kVals, d_kVals, numKs * sizeof (uint), 
                         hipMemcpyDeviceToHost)); 

    int kMaxIndex = numKs - 1;
    int kOffsetMax = 0;
    while (kVals[kMaxIndex] == length) {
      output[kIndices[numKs-1]] = maximum;
      numKs--;
      kMaxIndex--;
      kOffsetMax++;
    }

    int kOffsetMin = 0;
    while (kVals[0] == 1) {
      output[kIndices[0]] = minimum;
      kIndices++;
      kVals++;
      numKs--;
      kOffsetMin++;
    }

    timing(1,1);
    /// ***********************************************************
    /// **** STEP 2: CreateBuckets 
    /// ****  Declare and Generate Pivots and Slopes
    /// ***********************************************************
    timing(0,2);
    // since slopes and pivots will be reused as oldminimums and oldslopes, preallocate to the right size
    uint slopesize = max(numPivots, numKs);
    CUDA_CALL(hipMalloc(&d_slopes, slopesize * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_pivots, slopesize * sizeof(T)));
    CUDA_CALL(hipMalloc(&d_pivottree, numPivots * sizeof(T)));

    // Find bucket sizes using a randomized selection
    generatePivots<T>(pivots, pivottree, slopes, d_vector, length, numPivots, sampleSize, 
                      numBuckets, minimum, maximum);
    SAFEcuda("generatePivots");
    // make any slopes that were infinity due to division by zero (due to no 
    //  difference between the two associated pivots) into zero, so all the
    //  values which use that slope are projected into a single bucket
    for (register int i = 0; i < numPivots - 1; i++)
      if (isinf(slopes[i]))
        slopes[i] = 0;

    // for(int i=0;i<numPivots-1;i++){
    //   printf("\n 3 Slope: %lf \n",slopes[i]);
    // }
    CUDA_CALL(hipMemcpy(d_slopes, slopes, numPivots * sizeof(double), 
                         hipMemcpyHostToDevice));  
    CUDA_CALL(hipMemcpy(d_pivots, pivots, numPivots* sizeof(T), 
                         hipMemcpyHostToDevice)); 
    CUDA_CALL(hipMemcpy(d_pivottree, pivottree, numPivots* sizeof(T), 
                         hipMemcpyHostToDevice));
    timing(1,2);
    /// ***********************************************************
    /// **** STEP 3: AssignBuckets 
    /// **** Using the function assignSmartBucket
    /// ***********************************************************
    timing(0,3);

    //Distribute elements into their respective buckets
    assignSmartBucket<T><<<numBlocks, threadsPerBlock, 2 * numPivots * sizeof(T) +  
      + numPivots * sizeof(double) + numBuckets * sizeof(uint)>>>
      (d_vector, length, numBuckets, d_slopes, d_pivots, d_pivottree, numPivots, 
       d_elementToBucket, d_bucketCount, offset);
    SAFEcuda("assignSmartBucket");

    CUDA_CALL(hipMemcpy(slopes, d_slopes, numPivots * sizeof(double), 
                         hipMemcpyDeviceToHost));  


    timing(1,3);
    /// ***********************************************************
    /// **** STEP 4: IdentifyActiveBuckets 
    /// **** Find the kth buckets
    /// **** and update their respective indices
    /// ***********************************************************
    timing(0,4);

    sumCounts<<<numBuckets/threadsPerBlock, threadsPerBlock>>>(d_bucketCount, 
                                                               numBuckets, numBlocks);
    SAFEcuda("sumCounts");

    findKBuckets(d_bucketCount, h_bucketCount, numBuckets, kVals, numKs, 
                 kthBucketScanner, kthBuckets, numBlocks);
    SAFEcuda("findKBuckets");

    // we must update K since we have reduced the problem size to elements in the 
    // kth bucket.
    //  get the index of the first element
    //  add the number of elements
    uniqueBuckets[0] = kthBuckets[0];
    reindexCounter[0] = 0;
    numUniqueBuckets = 1;
    kVals[0] -= kthBucketScanner[0];

    for (int i = 1; i < numKs; i++) {
      if (kthBuckets[i] != kthBuckets[i-1]) {
        uniqueBuckets[numUniqueBuckets] = kthBuckets[i];
        reindexCounter[numUniqueBuckets] = 
          reindexCounter[numUniqueBuckets-1]  + h_bucketCount[kthBuckets[i-1]];
        numUniqueBuckets++;
      }
      kVals[i] = reindexCounter[numUniqueBuckets-1] + kVals[i] - kthBucketScanner[i];
    }

    newInputLength = reindexCounter[numUniqueBuckets-1] 
      + h_bucketCount[kthBuckets[numKs - 1]];
    
    // reindex the counts
    CUDA_CALL(hipMalloc(&d_reindexCounter, numKs * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_uniqueBuckets, numKs * sizeof(uint)));

    CUDA_CALL(hipMemcpy(d_reindexCounter, reindexCounter, 
                         numUniqueBuckets * sizeof(uint), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_uniqueBuckets, uniqueBuckets, 
                         numUniqueBuckets * sizeof(uint), hipMemcpyHostToDevice));

    reindexCounts<<<(int) ceil((float)numUniqueBuckets/threadsPerBlock), 
      threadsPerBlock>>>(d_bucketCount, numBuckets, numBlocks, d_reindexCounter, 
                         d_uniqueBuckets, numUniqueBuckets);
    SAFEcuda("reindexCounts");

    timing(1,4);
    /// ***********************************************************
    /// **** STEP 5: Reduce 
    /// **** Copy the elements from the unique acitve buckets
    /// **** to a new vector 
    /// ***********************************************************
    timing(0,5);

    // allocate memory for the new array
    CUDA_CALL(hipMalloc(&newInput, newInputLength * sizeof(T)));
   
    int numUnique_extended = ( 2 << (int)( floor( log2( (float)numUniqueBuckets ) ) ) );
    if (numUnique_extended > numUniqueBuckets+1){
      numUnique_extended--;
    } else {
      numUnique_extended = (numUnique_extended << 1 ) - 1;
    }


    copyElements_tree<T><<<numBlocks, threadsPerBlock, 
      numUnique_extended * sizeof(uint)>>>(d_vector, length, d_elementToBucket, 
                                           d_uniqueBuckets, numUniqueBuckets, numUnique_extended, newInput, offset, 
                                           d_bucketCount, numBuckets);
    SAFEcuda("copyElements");


    timing(1,5);
    /// ***********************************************************
    /// **** STEP 6: sort&choose
    /// **** Using thrust::sort on the reduced vector and the
    /// **** updated indices of the order statistics, 
    /// **** we solve the reduced problem.
    /// ***********************************************************
    timing(0,6);


    // declare and allocate device memory for recursion phase
    uint *d_numUniquePerBlock, *d_markedBucketFlags, *d_sums, *d_Kbounds, *d_oldReindexCounter;
    CUDA_CALL(hipMalloc(&d_numUniquePerBlock, (numKs) * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_markedBucketFlags, numKs * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_sums, numKs * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_Kbounds, (numKs+1) * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_oldReindexCounter, numKs * sizeof(uint)));

    // I'm not sure if new Minimums needs to be double or not.
    // if not, we can also declare old minimums and use d_pivots
    double * d_newMinimums;
    //    T * d_newMinimums;
    double * d_newSlopes, *d_oldSlopes;
    CUDA_CALL(hipMalloc(&d_newMinimums, numKs * sizeof(double)));
    //    CUDA_CALL(hipMalloc(&d_newMinimums, numNewActive * sizeof(T)));
    CUDA_CALL(hipMalloc(&d_newSlopes, numKs * sizeof(double)));
    d_oldSlopes = d_slopes;

    //    uint* tempReindex;
    int recreateBlocks;
    //		double * tempSlopes;

    // old minimums
    double * d_oldMinimums;
    CUDA_CALL(hipMalloc(&d_oldMinimums, slopesize * sizeof(double)));
    convertMinimums<T><<<1,1024>>>(d_oldMinimums, d_pivots, slopesize);

    CUDA_CALL(hipMalloc (&newInputAlt, sizeof(T) * newInputLength));


    CUDA_CALL(hipMemcpy (d_kVals, kVals, numKs * sizeof (uint), hipMemcpyHostToDevice));


    // declare and initialize parameters for recursion
    uint numOldActive, numNewActive, oldNumSmallBuckets, newNumSmallBuckets;
    numOldActive = numPivots - 1;
    numNewActive = numUniqueBuckets;
    oldNumSmallBuckets = numBuckets/numOldActive;


    int recreateThreads = 128;
    int test = 0;
    // *****************************************************
    // Here seems to be where we begin the recursion
    // *****************************************************
    while (newInputLength > numKs && test < 4){
      test++;
      //		printActive<<<1,1>>>(d_uniqueBuckets,numNewActive);

      /*
        printf("\n *** *** *** *** \t *** *** *** \t *** *** *** *** \n");

        printKVals<<<1,1>>>(d_kVals,numKs);
        printf("\n *** *** *** *** \n *** * Input * *** \n *** *** *** *** \n");
        printInput<T><<<1,1>>>(0, newInputLength, newInput);
        hipDeviceSynchronize();

        printf("\n *** *** *** *** \t *** *** *** \t *** *** *** *** \n");
      */


      // determine the number of buckets per new block
      newNumSmallBuckets = min(11264,numBlocks*numBuckets/numNewActive);
      //    newNumSmallBuckets = numBuckets/numNewActive;

      std::cout << "oldNumSmallBuckets = " << oldNumSmallBuckets << "     newNumSmallBuckets = " << newNumSmallBuckets << std::endl;

      printf("\n ******** \n iteration %d \n ******** \n",test);
      printf("\n ******** \n current elements %d \n ******** \n",newInputLength);

      recreateBlocks = (uint) ceil((float)numNewActive/recreateThreads);



      // Recreate sub-buckets
      recreateBuckets<T><<<recreateBlocks, recreateThreads
        , numOldActive*sizeof(double)*2>>>(d_uniqueBuckets, d_newSlopes, d_newMinimums
                                           , numNewActive, d_oldSlopes, d_oldMinimums, numOldActive
                                           , oldNumSmallBuckets, newNumSmallBuckets);


      hipDeviceSynchronize();
      SAFEcuda("recreateBuckets");

      
      if (test > 1) {
        //printf ("d_reindexCounter\n");
        //printNumUnique<<<numBlocks, threadsPerBlock>>>(d_reindexCounter, numNewActive);
        //printf ("d_numUniquePerBlock\n");
        //printNumUnique2<<<numBlocks, threadsPerBlock>>>(d_numUniquePerBlock, numNewActive);
        //printf ("d_KBounds\n");
        //printNumUnique<<<numBlocks, threadsPerBlock>>>(d_Kbounds, numNewActive);
        sortBlock<T><<<numOldActive, threadsPerBlock / 2
          , 100 * sizeof(T)>>>(newInput, newInputLength, d_reindexCounter, numOldActive, d_numUniquePerBlock, d_uniqueBuckets, d_oldMinimums, d_bucketCount, numKs, d_Kbounds); 
      }
      hipDeviceSynchronize();
      SAFEcuda("sortBlock");

      //printDeviceMemory_uint<<<1,1>>>(d_bucketCount,"dbCount",15);



      reassignBuckets<T><<<numNewActive, threadsPerBlock
        , newNumSmallBuckets*sizeof(uint)>>>(newInput, newInputLength, d_reindexCounter, d_newSlopes
                                             , d_newMinimums, numNewActive, newNumSmallBuckets
                                             , d_elementToBucket, d_bucketCount);

      //printDeviceMemory_uint<<<1,1>>>(d_bucketCount,"dbCount",15);
      hipDeviceSynchronize();
      hipDeviceSynchronize();
      SAFEcuda("reassignBuckets");


      //Update variables for recursion

      //    tempReindex = d_oldReindexCounter;
      //    d_oldReindexCounter = d_reindexCounter;
      //    d_reindexCounter = tempReindex;


      numOldActive = numNewActive;

      //std::cout << "old = " << d_oldSlopes << "  new = " << d_newSlopes << std::endl;
      pointerSwap<double>(&d_oldSlopes,&d_newSlopes);
      //std::cout << "old = " << d_oldSlopes << "  new = " << d_newSlopes << std::endl;


      oldNumSmallBuckets = newNumSmallBuckets;

      hipDeviceSynchronize();
    
      //    timing(1,6);
     
      SAFEcuda("pre findKBuckets");
      hipDeviceSynchronize();

      //CUDA_CALL(hipMemcpy(reindexCounter,d_oldReindexCounter,numKs * sizeof(uint),hipMemcpyDeviceToHost));


      // compute Kbounds and copy to device
      uint blockOffset;
      uint Kbounds[numKs+1];
      Kbounds[0]=0;
      uint j = 0;
      uint i = 1;
      uint k = kVals[j];
      while (i < numOldActive) {
        blockOffset = reindexCounter[i]; 
        while ( (k <= blockOffset) && (j < numKs) ) {
          j++;
          k = kVals[j];
        }
        Kbounds[i]=j;
        i++;
      }
      Kbounds[numOldActive]=numKs;

      CUDA_CALL(hipMemcpy(d_Kbounds, Kbounds, 
                           (numOldActive+1) * sizeof(uint), hipMemcpyHostToDevice));
      // *******************************
      hipDeviceSynchronize();


      pointerSwap<uint>(&d_oldReindexCounter,&d_reindexCounter);
      /*
        std::cout << "d_uniqueBuckets = " << d_uniqueBuckets << " d_newSlopes = " << d_newSlopes << std::endl;
        std::cout << "d_newMinimums = " << d_newMinimums << " d_oldSlopes = " << d_oldSlopes << " d_oldMinimums = " << d_oldMinimums << std::endl;
        std::cout << "d_reindexCounter = " << d_reindexCounter << std::endl;
        std::cout << "d_oldReindexCounter = " << d_oldReindexCounter << std::endl;
        std::cout << "numKs = " << numKs << std::endl;
        std::cout << "d_Kbounds = " << d_Kbounds << std::endl;

        std::cout <<  std::endl;
        std::cout << "numNewActive = " << numNewActive << std::endl;
      */

      printKVals<<<1,1>>>(d_kVals,numKs);

      newInputLengthAlt = findKbucketsByBlock (d_bucketCount, d_oldReindexCounter, d_Kbounds, d_reindexCounter, d_sums, d_kVals
                                               , d_markedBucketFlags, d_numUniquePerBlock, d_uniqueBuckets, &numNewActive
                                               , numOldActive, newNumSmallBuckets, numKs);
      SAFEcuda("findKBucketsByBlock");

      hipDeviceSynchronize();


      //printNumUnique<<<1,1>>>(d_numUniquePerBlock, numKs);

      //	printMinimums<<<1,1>>>(d_oldMinimums, numKs);

      //printMinimums<<<1,1>>>(d_newMinimums, numKs);

      printNumUnique<<<1,1>>>(d_numUniquePerBlock,numKs);


      pointerSwap<double>(&d_newMinimums,&d_oldMinimums);

      int reducedlength = Reduction<T>(newInput, newInputAlt, d_elementToBucket, d_uniqueBuckets, d_oldReindexCounter, d_numUniquePerBlock, newInputLength, numOldActive, numNewActive);
      SAFEcuda("Reduction");

      //printKVals<<<1,1>>>(d_kVals,numKs);


      hipDeviceSynchronize();

      if (newInputLength < 600){
        printf("\n *** *** *** *** \n *** * Input * *** \n *** *** *** *** \n");
        printInput<T><<<1,1>>>(0, newInputLength, newInput);
        hipDeviceSynchronize();
        printf("\n *** *** *** *** \n *** * Alt * *** \n *** *** *** *** \n");
        printInput<T><<<1,1>>>(0, newInputLengthAlt, newInputAlt);
        hipDeviceSynchronize();
        printf("\n *** *** *** *** \n *** * OldReindex * *** \n *** *** *** *** \n");
        printReindex<<<1,1>>>(d_oldReindexCounter, numOldActive);
        hipDeviceSynchronize();
        printf("\n *** *** *** *** \n *** * Reindex * *** \n *** *** *** *** \n");
        printReindex<<<1,1>>>(d_reindexCounter, numNewActive);
        hipDeviceSynchronize();
        printFlag<<<1,1>>>(d_elementToBucket, newInputLength);
        hipDeviceSynchronize();
      }

      /*
        printf("\n ***** New *****\n");
        printReindex<<<1,1>>>(d_reindexCounter, numKs);

        if (newInputLengthAlt < reducedlength) {
        printInput<T><<<1,1>>>(newInputLengthAlt, reducedlength, newInputAlt);
        }
      */

      /*
        correctBuckets<T><<<numOldActive,1024>>>(d_numUniquePerBlock, newInput, newInputAlt, d_oldReindexCounter, numOldActive, numNewActive, d_newMinimums);
        SAFEcuda("correctBuckets");
        hipDeviceSynchronize();
      */

      checkBuckets(newInputAlt, Kbounds, reindexCounter, numOldActive, numKs, newInputLengthAlt, numNewActive);
      /*
        for(int i = 0; i < numKs; i++){
        printf("checkBuckets reindexCounter[%d] = %d\n",i,reindexCounter[i]);
        }
      */







      CUDA_CALL(hipMemcpy(reindexCounter, d_reindexCounter, 
                           numKs * sizeof(uint), hipMemcpyDeviceToHost));


      std::cout << "NewInputLengthAlt = " << newInputLengthAlt << "     reducedlength = " << reducedlength << std::endl;



      std::cout << "numNewActive = " << numNewActive << std::endl;
      std::cout << "numOldActive = " << numOldActive << std::endl;
      std::cout <<  std::endl;
		
      hipGetLastError();

      CUDA_CALL(hipMemcpy (kVals, d_kVals, numKs * sizeof (uint), hipMemcpyDeviceToHost));
      //    CUDA_CALL(hipMemcpy(uniqueBuckets, d_uniqueBuckets, 
      //                        numNewActive * sizeof(uint), hipMemcpyDeviceToHost));
      /*    CUDA_CALL(hipMemcpy(reindexCounter, d_reindexCounter, 
            numKs * sizeof(uint), hipMemcpyDeviceToHost));


            checkBuckets(newInputAlt, Kbounds, reindexCounter, numNewActive);
      */
      newInputLength = newInputLengthAlt;

      pointerSwap<T>(&newInput,&newInputAlt);

      hipDeviceSynchronize();
      /*
        printf("INPUT %d\n *********\n",test);
        printInput<T><<<1,1>>>(newInputLength,newInput);
      */

    }

    hipFree(d_pivots);
    hipFree(d_pivottree);
    free(h_bucketCount);
    hipFree(d_bucketCount);
    hipFree(d_uniqueBuckets); 
    hipFree(d_markedBucketFlags); 
    hipFree(d_sums); 
    hipFree(d_reindexCounter);
    hipFree(d_oldReindexCounter);
    hipFree(d_newMinimums);
    hipFree(d_oldMinimums);
    hipFree(d_numUniquePerBlock);
    hipFree(d_Kbounds);
    hipFree(d_oldSlopes); 
    hipFree(d_newSlopes);

    // OLD STUFF BEGINS
    timing(1,6);
    timing(0,7);
    //printf ("NEW LENGTH = %d\n", newInputLengthAlt);

    sort_phase<T>(newInput, newInputLength);
    //    sort_phase<T>(newInputAlt, newInputLengthAlt);
    SAFEcuda("sort_phase");

    T * d_output = (T *) d_elementToBucket;
    CUDA_CALL(hipMemcpy (d_output, output, 
                          (numKs + kOffsetMin + kOffsetMax) * sizeof (T), 
                          hipMemcpyHostToDevice));

    CUDA_CALL(hipMemcpy (d_kIndices, kIndices, numKs * sizeof (uint), 
                          hipMemcpyHostToDevice));

    copyValuesInChunk<T><<<numBlocks, threadsPerBlock>>>(d_output, newInput, d_kVals, d_kIndices, numKs);
    //    copyValuesInChunk<T><<<numBlocks, threadsPerBlock>>>(d_output, newInputAlt, d_kVals, d_kIndices, numKs);
    SAFEcuda("copyValuesInChunk");

    CUDA_CALL(hipMemcpy (output, d_output, 
                          (numKs + kOffsetMin + kOffsetMax) * sizeof (T), 
                          hipMemcpyDeviceToHost));

    //free all used memory
 
 
  
    hipFree(d_elementToBucket);  
    hipFree(d_kIndices); 
    hipFree(d_kVals); 
    hipFree(newInput); 
    hipFree(newInputAlt);
    free (kIndices - kOffsetMin);


  


    timing(1,7);
    return 1;
  }


  /* Wrapper function around the multi-selection fucntion that inverts the given k indices.
   */
  template <typename T>
  T bucketMultiselectWrapper (T * d_vector, int length, uint * kVals_ori, int numKs
                              , T * outputs, int blocks, int threads) { 

    int numBuckets = 8192;
    uint kVals[numKs];

    // turn it into kth smallest
    for (register int i = 0; i < numKs; i++) 
      kVals[i] = length - kVals_ori[i] + 1;
   
    bucketMultiSelect<T> (d_vector, length, kVals, numKs, outputs, blocks, threads, numBuckets, 17);

    return 1;
  }
}

