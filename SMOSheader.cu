/* Copyright 2012 Jeffrey Blanchard, Erik Opavsky, and Emircan Uysaler
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//#define GPUNUMBER 2

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <unistd.h>

#include <algorithm>
//Include various thrust items that are used
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/random.h>

//Include CUB and MGPU sorting
#include "cubdevicesort.cu"
#include "mgpudevicesort.cu"

//various functions, include the functions
//that print numbers in binary.
#include "printFunctions.cu"

//the algorithms
#include "bucketMultiselect.cu"
#include "bucketMultiselect_thrust.cu"
#include "bucketMultiselect_cub.cu"
#include "bucketMultiselect_mgpu.cu"
#include "naiveBucketMultiselect.cu"

#include "generateProblems.cu"
#include "multiselectTimingFunctions.cu"

