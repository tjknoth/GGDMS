#include "hip/hip_runtime.h"
// -*- c++ -*-  

template<typename T>
void mmsetToAllZero (T * d_vector, int length) {
  hipMemset(d_vector, 0, length * sizeof(T));
}

void printDevice(uint * d_reindexSums, uint * d_sums, int numKs){
  for (int i = 0; i < numKs; i ++){
    printf("reindexSum[%d] = %u \t sum[%d] = %u \n",i,d_reindexSums[i],i,d_sums[i]);
  }
}


/*
 * This device function will be called by a kernel with each thread finding the 
 * k buckets for each oldActiveBucket (i.e. each block used to reassign buckets) 
 */

__global__ void	printActive(uint * d_uniqueBuckets, int numActive){
  if(threadIdx.x + blockIdx.x == 0) {
    printf("\n ************************ \n");
    printf("numNewActive: %d",numActive);
    for (int i = 0; i < numActive; i ++){
      printf("uniqueBuckets[%d] = %u\n",i,d_uniqueBuckets[i]);
    }
    printf("\n ************************ \n");
  }
}

/*
  __device__ void correctBlocks(int blockNumKs){

  if(blockNumKs > 1) {
  int i = 0;
  int j = 0;
		
  }

  }
*/


__device__ uint d_findKBucketsByBlock(uint * d_bucketCount, uint * kVals, uint * markedBuckets, 
                                      uint * sums, uint * reindexsums, const int numNewSmallBuckets, 
                                      const int blockBucketOffset, const int blockStart,  
                                      const int blockNumKs, const int blockKsOffset, uint * markedBucketFlags)
{
  int kBucket = blockBucketOffset;
  int blockMaxBucket = blockBucketOffset + numNewSmallBuckets;
  int k;
  int sum = blockStart;
  uint temp;
  int numUniqueBlock=1;
  markedBucketFlags[blockKsOffset] = 1;


  // find the buckets which contain the kVals
  for(register int i = 0; i < blockNumKs; i++) {
    k = kVals[blockKsOffset + i];
    while ((sum < k) & (kBucket < blockMaxBucket)) {
      temp = d_bucketCount[kBucket];
      sum += temp;     
      kBucket++;
    } // end while
    markedBuckets[blockKsOffset + i] = kBucket-1;
    //printf("markedBuckets[%d] = %d\n",blockKsOffset + i,markedBuckets[blockKsOffset + i]);
    sums[blockKsOffset + i] = sum - temp; 
    reindexsums[blockKsOffset + i] = temp; 

    // determine if this marked bucket is unique
    //  if so, increase the unique counter and create a flag for uniqueness
    //  if not, ensure the flag is 0 and remove the count from reindexsums 
    //          in order to have an accurate cummulative sum outside this kernel
    if (i>0) {
      if ( (markedBuckets[blockKsOffset + i] != markedBuckets[blockKsOffset + i - 1]) ) {
        numUniqueBlock++;
        markedBucketFlags[blockKsOffset + i] = 1;
      } else { 
        markedBucketFlags[blockKsOffset + i] = 0;
        reindexsums[blockKsOffset + i - 1] = 0; 
      }  // end if-else
    } // if buckets are equal

  } // end for

  return numUniqueBlock;

} // end device function d_findKBucketsByBlock

/*
 * The kernel will launch one thread per old active bucket to get the new k buckets
 * which should be identified as active.  It also gets the sum of all elements that 
 * were in previous new buckets in order to update the desired order statistics.
 * Launch this kernel with a fixed number of threads per block, probably 64, and enough 
 * blocks to achieve numOldActive total threads.
 * No shared memory required.
 */
__global__ void findKbucketsByBlock_kernel (uint * d_bucketCount, uint * d_kVals, uint * d_markedBuckets, uint * d_sums, uint * d_reindexsums, uint * d_bucketBounds, uint * d_KBounds, const int numNewSmallBuckets, const int numOldActive, const int numKs, uint * numUniquePerBlock, uint * markedBucketFlags)
{

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int blockKsOffset, blockNumKs;
  /*
    if (index < numOldActive + 1) {
    deviceTag(2);
    blockKsOffset = d_KBounds[index];
    //printf("blockKsOffset %d\n",blockKsOffset);			
    blockNumKs = d_KBounds[index+1] - blockKsOffset;
  */
  if (index < numOldActive) {

    blockKsOffset = d_KBounds[index];
    //printf("blockKsOffset %d\n",blockKsOffset);
    if (index + 1 < numOldActive) {			
      blockNumKs = d_KBounds[index+1] - blockKsOffset;

    } else {
      blockNumKs = numKs - blockKsOffset;

    }
    if (blockNumKs > 1) {
      printf("index = %d \t blockNumKs = %d\n",index,blockNumKs);
      printf("bucketBounds[%d] = %u \t bucketBounds[%d] = %u \n",index,d_bucketBounds[index],index + 1,d_bucketBounds[index + 1]);
    }

    syncthreads();
    /*
      if (index+1 == numOldActive) {

      //				blockKsOffset = d_KBounds[index];
      blockNumKs = numKs - blockKsOffset;  // potentially unnecessary based on kBounds
      }
    */
    if (blockNumKs > 0){
      numUniquePerBlock[index]=d_findKBucketsByBlock ( d_bucketCount, d_kVals, d_markedBuckets, d_sums, d_reindexsums, numNewSmallBuckets, numNewSmallBuckets*index, d_bucketBounds[index], blockNumKs, blockKsOffset, markedBucketFlags);
    } else {
      numUniquePerBlock[index]=0;
    }  
  } // end if index
} // end kernel findKbucketByBlock_kernel

/*
 * This kernel updates the order statistics kVals by block.
 * It should be launched with one block per old active bucket and max threads 
 */
__global__ void updateKValsByBlock (uint * d_kVals, uint * d_sums, uint * d_reindexsums, uint * d_KBounds, const int numKs, const uint numOldActive)
{
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int index;
  __shared__ uint blockKsOffset;
  __shared__ uint numKsInBlock;

  uint blockKupdate;
  if (blockId<numOldActive) {

    if (threadId<1){
      blockKsOffset = d_KBounds[blockId];
      numKsInBlock = d_KBounds[blockId+1]-blockKsOffset;
    } // end if threadId<1

    syncthreads();

    //     if (threadId<numKsInBlock){
    for (index=threadId; index<numKsInBlock; index+=blockDim.x){
      blockKupdate = d_reindexsums[blockKsOffset + threadId] - d_sums[blockKsOffset + threadId];
      d_kVals[blockKsOffset + threadId] += blockKupdate;
    } // end threadId<numKsInBlock

  } // end if (blockId<numOldActive)

} // end kernel updateKValsByBlock

   



// the host function to find k buckets by block
inline int findKbucketsByBlock (uint * d_bucketCount, uint * d_bucketBounds, uint * d_Kbounds, 
                                uint * d_reindexsums, uint * d_sums, uint * d_kVals, uint * d_markedBucketFlags
                                , uint * d_numUniquePerBlock, uint *d_uniqueBuckets, uint * numUniqueBuckets
                                , const uint numOldActive, const uint numNewSmallBuckets, const uint numKs)
{
  uint h_numUnique, h_lastActiveCount, h_newActivePrefix;
  uint *numSelected, *numSelectedSum;
  CUDA_CALL(hipMalloc(&numSelected, sizeof(uint)));
  CUDA_CALL(hipMalloc(&numSelectedSum, sizeof(uint)));


  // set threads and compute numBlocks 
  int numFindThreads = 64;
  int numFindBlocks = (int) ceil((float)numOldActive/numFindThreads);
   
  // launch the kernel
  findKbucketsByBlock_kernel<<<numFindBlocks,numFindThreads>>>(d_bucketCount, d_kVals, d_uniqueBuckets, d_sums, d_reindexsums
                                                               , d_bucketBounds, d_Kbounds, numNewSmallBuckets, numOldActive
                                                               , numKs,d_numUniquePerBlock, d_markedBucketFlags);

  // get the count of the last active buckeet
  CUDA_CALL(hipMemcpy(&h_lastActiveCount, d_reindexsums+numKs-1, sizeof(uint), hipMemcpyDeviceToHost));

  // compute a cumulative sum of reindex sums which currently contains the bucket counts for all marked buckets
  cubDeviceExclusiveSum<uint>(d_reindexsums,d_reindexsums,numKs);

  // update the kVals using the offsets from the old buckets (d_sums) and the new buckets (d_reindexsums)
  updateKValsByBlock<<<numOldActive,MAX_THREADS_PER_BLOCK>>>(d_kVals, d_sums, d_reindexsums, d_Kbounds, numKs, numOldActive);

  // use the flags identifying unique buckets (d_markedBucketFlags) to extra a unique list of active buckets
  // and to extract the appropriate cont offsets for the new data vector 
  SelectFlagged(d_uniqueBuckets,d_uniqueBuckets,d_reindexsums,d_reindexsums,d_markedBucketFlags,numSelected,numKs);
 
  // compute an exlusive sum of the unique buckets per block to obtain a bucket offseet list

  CUDA_CALL(hipMemcpy(&h_numUnique, numSelected, sizeof(uint), hipMemcpyDeviceToHost));

  cubDeviceExclusiveSum<uint>(d_numUniquePerBlock,d_numUniquePerBlock,h_numUnique);
 
  // obtain the cummulative count of elements in new active buckets (excluding the last unique bucket)
  CUDA_CALL(hipMemcpy(&h_newActivePrefix, d_reindexsums+h_numUnique-1, sizeof(uint), hipMemcpyDeviceToHost));

  // obtain the full length of the new data and the number of unique buckets
  uint newInputLength = h_newActivePrefix + h_lastActiveCount;
  *numUniqueBuckets = h_numUnique;

  //cleanup
  hipFree(numSelected);
  hipFree(numSelectedSum);

  return newInputLength;

} // end findKbucketsByBlock host function



__global__ void ReductionFlags (uint * d_elementToBucket, uint* d_uniqueBuckets, uint* d_blockBounds, uint *d_uniqueBounds, const uint numElements, const uint numOldActive, const uint numNewActive)
{
  __shared__ uint blockOffset;
  __shared__ uint blockUniqueOffset;
  __shared__ uint numElementsInBlock;
  __shared__ uint numUniqueInBlock;

  extern __shared__ uint activeBuckets[];

  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int index;
  uint temp, low, mid, high, compare;
  if (blockId < numOldActive) {
    if (threadId < 1) {
      blockOffset = d_blockBounds[blockId];
      blockUniqueOffset = d_uniqueBounds[blockId];
      if (blockId +1 < numOldActive) {
        numElementsInBlock = d_blockBounds[blockId+1]-blockOffset;
        numUniqueInBlock = d_uniqueBounds[blockId+1]-blockUniqueOffset;
      } else {
        numElementsInBlock = numElements-blockOffset;
        numUniqueInBlock = numNewActive-blockUniqueOffset;
      }
    }

    syncthreads();

    // read unique active buckets into shared memory
    for (int i = threadId; i < numUniqueInBlock; i += blockDim.x) {
      activeBuckets[i] = d_uniqueBuckets[blockUniqueOffset + i];
    }
    activeBuckets[numUniqueInBlock]=2200000; // ensure the binary search does not incorrectly identify a bucket as active


    syncthreads();

    // binary search the active buckets for this block
    for (int i = threadId; i < numElementsInBlock; i += blockDim.x) {
      index = blockOffset + i;
      temp = d_elementToBucket[index];
      low = 0;
      high = numUniqueInBlock;
      compare = 0;  
      for (int j = 1; j < numUniqueInBlock + 1; j *= 2) {
        mid = (high + low) / 2;
        compare = (temp > activeBuckets[mid]);
        low = compare ? mid : low;
        high = compare ? high : mid;
      } //end for

      // if the current bucket is active, flag as 1 otherwise flag as 0 using elementToBucket as flag vector
      if (temp==activeBuckets[high]) {
        d_elementToBucket[index]=1;
        //       d_elementToBucket[index] = numUniquePerBlock;
      } else {
        d_elementToBucket[index]=0;
      } // end if else temp==active

    } // end if threadID< numElementsInBlock
  } // end if blockId<numOldActive
} // end kernel ReductionFlags
    

  
template<typename T>
int Reduction(T* d_vec, T* d_new, uint * d_elementToBucket, uint* d_uniqueBuckets, uint* d_blockBounds, uint *d_uniqueBounds, const uint numElements, const uint numOldActive, const uint numNewActive)
{  
  uint *numSelected;
  uint h_numSelected;
  CUDA_CALL(hipMalloc(&numSelected, sizeof(uint)));

  // identify the active buckets by block and mark them with flags in d_elementToBucket  
  ReductionFlags<<<numOldActive,MAX_THREADS_PER_BLOCK,numOldActive*sizeof(uint)>>>(d_elementToBucket, d_uniqueBuckets, d_blockBounds, d_uniqueBounds, numElements, numOldActive, numNewActive);

  // use the flags from the previous kernel select the elements from d_vec
  cubDeviceSelectFlagged<T>(d_vec, d_elementToBucket, d_new, numSelected, numElements);

  CUDA_CALL(hipMemcpy(&h_numSelected, numSelected, sizeof(uint), hipMemcpyDeviceToHost));

  hipFree(numSelected);

  return h_numSelected;
}  // end function Reduction


template <typename T>
__global__ void printStatement(T * newInput, T * newInputAlt, int newInputLength, int newInputLengthAlt, int newNumSmallBuckets, int numNewActive, uint * d_oldReindexCounter, uint * d_reindexCounter){
  if (threadIdx.x + blockIdx.x == 0){

    /*
      for (int i = 0; i < 3; i+=2) {
      printf("newInput %d = %.10lf\t newInput %d = %.10lf\n",
      i,newInput[i],i+1,newInput[i+1]);
      printf("\n");
      } 

      for (int i = 0; i < 3; i+=2) {
      printf("newInputAlt %d = %lf\t newInputAlt %d = %lf\n",
      i,newInputAlt[i],i+1,newInputAlt[i+1]);
      printf("\n");
      }
    */

    printf("newInputLength %d \t newInputLengthAlt %d\n",newInputLength,newInputLengthAlt);
    printf("newNumSmallBuckets %d\n",newNumSmallBuckets);
    printf("numNewActive %d\n",numNewActive);

  }
}


template <typename T>
__global__ void convertMinimums(double * d_oldMinimums, T * d_pivots, int slopesize){

  for (int i = threadIdx.x; i < slopesize; i+= blockDim.x) {
    d_oldMinimums[i] = (double) d_pivots[i];
  }

}




// *****************************
template <typename T>
__global__ void printDeviceMemory(T* d_variable, const char *variablestring, int length){

  for (int i = threadIdx.x; i < length; i += blockDim.x) {
    printf("%s[%d] =  %f\n",variablestring,i,d_variable[i]);
  }

}

/*
  template < >
  __global__ void printDeviceMemory < uint > (uint* d_variable, const char *variablestring, int length){

  for (int i = threadIdx.x; i < length; i += blockDim.x) {
  printf("%s[%d] =  %u\n",variablestring,i,d_variable[i]);
  }
  }
*/

__global__ void printDeviceMemory_uint (uint* d_variable, const char *variablestring, int length){

  for (int i = threadIdx.x; i < length; i += blockDim.x) {
    printf("%s[%d] =  %u\n",variablestring,i,d_variable[i]);
  }
}

// *****************************


__global__ void printSlopes(double* d_newSlopes,double* d_oldSlopes, int numKs){

  for (int i = threadIdx.x; i < numKs; i += blockDim.x) {
    printf("newSlope %d = %lf \t oldSlope %d = %lf \n",i,d_newSlopes[i],i,d_oldSlopes[i]);
  }

}

__global__ void printReindex(uint* d_reindexCounter, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    for(int i = 0; i < numKs; i+=2){
      printf("reindexCounter[%d] = %d\t",i,d_reindexCounter[i]);
      printf("reindexCounter[%d] = %d\n",i+1,d_reindexCounter[i+1]);
    }
  }
}


__global__ void printNumUnique(uint* d_numUnique, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    //int count = 0;
    for(int i = 0; i < numKs; i++){
      printf("numUnique[%d] = %d\n",i,d_numUnique[i]);
      //d_numUnique[i] = d_numUnique[i] - i - count;
      //if (d_numUnique[i] > count) count = d_numUnique[i];
    }
  }
}

__global__ void printNumUnique2(uint* d_numUnique, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    //int count = 0;
    for(int i = 0; i < numKs; i++){
      printf("numUniquePerBlock[%d] = %d\n",i,d_numUnique[i]);
      //d_numUnique[i] = d_numUnique[i] - i - count;
      //if (d_numUnique[i] > count) count = d_numUnique[i];
    }
  }
}

__global__ void multiBuckets(uint* d_numUnique, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    int count = 0;
    for(int i = 0; i < numKs; i++){
      printf("numUnique[%d] = %d\n",i,d_numUnique[i]);
      d_numUnique[i] = d_numUnique[i] - i - count;
      if (d_numUnique[i] > count) count = d_numUnique[i];
    }
  }
}


__global__ void printBucketCount(uint* d_bucketCount, int length){
  if (threadIdx.x + blockIdx.x == 0){
    int sum = 0;
    for(int i = 0; i < length; i++){
      //printf("bucketCount[%d] = %d\n",i,d_bucketCount[i]);
      sum += d_bucketCount[i];
    }
    printf("sum = %d\n",sum);
  }
}

__global__ void printKbounds(uint* d_Kbounds, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    for(int i = 0; i < numKs; i++){
      printf("Kbounds[%d] = %d\n",i,d_Kbounds[i]);
    }
  }
}

__global__ void printKVals(uint* d_kVals, int numKs){
  if (threadIdx.x + blockIdx.x == 0){
    for(int i = 0; i < numKs; i+=3){
      printf("kVals[%d] = %d\tkVals[%d] = %d\tkVals[%d] = %d\n",i,d_kVals[i],i+1,d_kVals[i+1],i+2,d_kVals[i+2]);
    }
  }
}

template <typename T>
void pointerSwap(T** pointer_a, T** pointer_b){
  T * tempPointer = * pointer_b;
  * pointer_b = * pointer_a;
  * pointer_a = tempPointer;
}

void doublePointerSwap(double** pointer_a, double** pointer_b){
  double * tempPointer = * pointer_b;
  * pointer_b = * pointer_a;
  * pointer_a = tempPointer;
}

void uintPointerSwap(uint** pointer_a, uint** pointer_b){
  uint * tempPointer = * pointer_b;
  * pointer_b = * pointer_a;
  * pointer_a = tempPointer;
}

void tag(int marker, int iteration){
  printf("%d Tag %d\n",iteration,marker);
}


template <typename T>
__global__ void printBuckets(int newInputLength, uint * d_elementToBucket, double * d_newMinimums, T * newInput) {

  for (int i = threadIdx.x; i < newInputLength; i += blockDim.x) {
    if (newInput[i] < d_newMinimums[i]) {
      printf("Element: %lf \t Minimum: %lf \n",newInput[i], d_newMinimums[i]);
    }
  }

}



__global__ void printFlag(uint * d_elementToBucket, int length) {

  for (int i = threadIdx.x; i < length; i += blockDim.x) {
    printf("elementToBucket[%d] = %u\n",i,d_elementToBucket[i]);

  }

}


template <typename T>
__global__ void printInput(int start, int end, T * newInput) {

  for (int i = start; i < end; i+=2) {
    printf("newInput[%d] = %.15lf \t",i,newInput[i]);
    printf("newInput[%d] = %.15lf \n",i+1,newInput[i+1]);
  }
}

/* Function to partition a block with multiple active buckets by active bucket
 */
template  <typename T>
__global__ void sortBlock(T* d_vec, int length, uint* d_bucketBounds, uint numBlocks, uint* numUniquePerBlock
                          , uint* uniqueBuckets, double* minimums, uint* d_bucketCount, uint numKs, uint* d_Kbounds) {
  int blockId = blockIdx.x;
  int numKsPerBlock = (blockId < numBlocks) ? (numUniquePerBlock[blockId + 1] - numUniquePerBlock[blockId]) : numKs - numUniquePerBlock[numBlocks];
  //int altNumKsPerBlock = (blockId < numBlocks - 1) ? (d_Kbounds[blockId + 1] - d_Kbounds[blockId]) : numKs - d_Kbounds[numBlocks - 1]; 

  if (numKsPerBlock > 1) { //|| altNumKsPerBlock > 1) {
    int threadId = threadIdx.x;
    int blockOffset = numUniquePerBlock[blockId];
    int firstBucket = uniqueBuckets[blockOffset];
    int blockLength = 0;
    for (int i = 0; i < numKsPerBlock; i++) {
      blockLength += d_bucketCount[uniqueBuckets[blockOffset + i]];
      //printf ("added %d from %d\n", d_bucketCount[uniqueBuckets[blockOffset + i]], uniqueBuckets[blockOffset + i]);
    }
    int blockStart = d_bucketBounds[blockOffset] - 1;
    if (threadId < 1)
      printf ("block = %d, blockLength = %d, firstBucket = %d, blockOffset = %d, blockStart = %d, numKs = %d\n", blockId, blockLength, firstBucket, blockOffset, blockStart, numKsPerBlock); 
    extern __shared__ uint array[];
    uint* offsets = (uint*) array;
    T* sharedVec = (T*) &offsets[numKsPerBlock];
    syncthreads();
    for (int i = threadId; i < blockLength; i += blockDim.x) {
      sharedVec[i] = d_vec[i + blockStart];
      printf ("copied %lf from index %d, i = %d, blockLength = %d\n", d_vec[i + blockStart], i + blockStart, i, blockLength);
    }
    syncthreads();
    if (threadId < 1) {
      //printf ("blockId = %d, numKsPerBlock = %d\n", blockId, numKsPerBlock);
      //printf ("firstBucket = %d, blockStart = %d, blockEnd = %d, blockId = %d\n", firstBucket, blockStart, blockEnd, blockId);
      offsets[0] = 0;
      for (int i = 1; i < numKsPerBlock; i++) {
        offsets[i] = d_bucketCount[firstBucket + i] + offsets[i - 1];
        //printf ("offsets[%d] = %d, added %d from block %d\n", i, offsets[i], d_bucketCount[firstBucket+i], blockId);
      }
      printf ("PARTITIONING, numKsPerBlock = %d, blockLength = %d, firstBucket = %d\n", numKsPerBlock, blockLength, firstBucket);
      for (int i = 0; i < blockLength; i++) {
        int j;
        int val = sharedVec[i];
        for (j = numKsPerBlock; minimums[j + blockOffset] > val && j > 0; j--);
        printf ("MINIMUM = %lf\n", minimums[j + blockOffset]);
        d_vec[i + blockStart] = sharedVec[i];
        printf ("COPIED back %f to %d on block %d\n", sharedVec[i], i + blockStart, blockId);
        offsets[j]++;
      } // end for
    } // end if (threadId < 1)

  } // end if
} // end sortBlock kernel

