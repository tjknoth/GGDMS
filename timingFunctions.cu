/* Copyright 2011 Russel Steinbach, Jeffrey Blanchard, Bradley Gordon,
 *   and Toluwaloju Alabi
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
template <typename T>
 struct results_t{
  float time;
  T val;
};

template<typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T* h_vec, uint size, results_t<T> **result){
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *)malloc(sizeof(results_t<T>));
}

template<typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, T* d_vec, results_t<T> *result, float time, T value){
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->val = value;
  result->time = time;
  //   hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////
//          THE SORT AND CHOOSE TIMING FUNCTION
/////////////////////////////////////////////////////////////////
template<typename T>
results_t<T>* timeSortAndChoose(T *h_vec, uint numElements, uint k){

  
  T* d_vec;
  T returnValueFromSelect;
  results_t<T> *result;
  float time;
  hipEvent_t start, stop;
 
  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);

  thrust::device_ptr<T> dev_ptr(d_vec);
  hipEventRecord(start, 0);

  thrust::sort(dev_ptr, dev_ptr + numElements);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  hipMemcpy(h_vec, d_vec, numElements * sizeof(T), hipMemcpyDeviceToHost);
  returnValueFromSelect = h_vec[numElements - k];

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);
  return result;
}



/////////////////////////////////////////////////////////////////
//          THE RADIX SELECT TIMING FUNCTION
/////////////////////////////////////////////////////////////////

template<typename T>
results_t<T>* timeRadixSelect(T *h_vec, uint numElements, uint k){


  float time;
  hipEvent_t start,stop;
  results_t<T> *result;
  T returnValueFromSelect;
  T *d_vec;

  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);
  thrust::device_ptr<T> dev_ptr(d_vec);

  hipEventRecord(start,0);
  //CALL THE WRAPPER FUNCTION
  returnValueFromSelect =  RadixSelect::RadixSelectWrapper(d_vec,numElements,k);
    
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);

  return result;
}
template<typename T>
results_t<T>* timeInplaceRadixSelect(T *h_vec, uint numElements, uint k){


  float time;
  hipEvent_t start,stop;
  results_t<T> *result;
  T returnValueFromSelect;
  T *d_vec;

  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);
  thrust::device_ptr<T> dev_ptr(d_vec);

  hipEventRecord(start,0);
  //CALL THE WRAPPER FUNCTION
  returnValueFromSelect =  InplaceRadix::inplaceRadixSelectWrapper(d_vec,numElements,k);
    
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);

  return result;
}

// /////////////////////////////////////////////////////////////////
// //          THE SLICING TIMING FUNCTION
// /////////////////////////////////////////////////////////////////

template<typename T>
results_t<T>* timeCuttingPlane(T *h_vec, uint size, uint k) {
  float time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  results_t<T> *result;
  T returnValueFromSelect;
  result = (results_t<T> *)malloc(sizeof(results_t<T>));

  thrust::device_vector<T> DataD(size);
  thrust::copy(h_vec,h_vec + size,DataD.begin());

  hipEventRecord(start,0);

  /* 
     UNCOMMENT TO GET ALGORITHM WORKING
     returnValueFromSelect = cp_select::median_min(DataD.begin(), DataD.end(),  7,  1, 1 + size - k);
  */
  returnValueFromSelect = NULL;

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->val =returnValueFromSelect;
  result->time = time;

  DataD.resize(0);
  return result;
  
}

//FUNCTION TO TIME BUCKET SELECT
template<typename T>
results_t<T>* timeBucketSelect(T* hostVec, uint size, uint k){
  hipEvent_t start, stop;
  float time;
  results_t<T> *result;
  T retFromSelect;
  T* deviceVec;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp,0);


  setupForTiming(start,stop, &deviceVec, hostVec, size, &result);

  hipEventRecord(start, 0);

  retFromSelect = BucketSelect::bucketSelectWrapper(deviceVec, size, k, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);


  wrapupForTiming(start,stop, deviceVec, result, time, retFromSelect);
  return result;

}

//FUNCTION TO TIME RANDOMIZED BUCKET SELECT
template<typename T>
results_t<T>* timeRandomizedBucketSelect(T* hostVec, uint size, uint k){
  hipEvent_t start, stop;
  float time;
  results_t<T> *result;
  T retFromSelect;
  T* deviceVec;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp,0);


  setupForTiming(start,stop, &deviceVec, hostVec, size, &result);

  hipEventRecord(start, 0);

  retFromSelect = RandomizedBucketSelect::randomizedBucketSelectWrapper(deviceVec, size, k, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);

  wrapupForTiming(start,stop, deviceVec, result, time, retFromSelect);
  return result;

}

//FUNCTION TO TIME NO EXTREMA RANDOMIZED BUCKET SELECT
template<typename T>
results_t<T>* timeNoExtremaRandomizedBucketSelect(T* hostVec, uint size, uint k){
  hipEvent_t start, stop;
  float time;
  results_t<T> *result;
  T retFromSelect;
  T* deviceVec;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp,0);


  setupForTiming(start,stop, &deviceVec, hostVec, size, &result);

  hipEventRecord(start, 0);

  retFromSelect = NoExtremaRandomizedBucketSelect::noExtremaRandomizedBucketSelectWrapper(deviceVec, size, k, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);

  wrapupForTiming(start,stop, deviceVec, result, time, retFromSelect);
  return result;

}


//FUNCTION TO TIME LANL SELECT
template<typename T>
results_t<T>* timeRandomizedSelect(T* hostVec, uint size, uint k)
{
  hipEvent_t start, stop;
  float time;
  results_t<T> *result;
  T returnValueFromSelect;
  T* deviceVec;

  setupForTiming(start,stop, &deviceVec, hostVec, size, &result);

  hipEventRecord(start, 0);

  returnValueFromSelect = randomizedSelect(deviceVec,size,k,.90);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);


  wrapupForTiming(start,stop, deviceVec, result, time,returnValueFromSelect);
  return result;

}
