/* Copyright 2012 Jeffrey Blanchard, Erik Opavsky, and Emircan Uysaler
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "SMOSheader.cu"

#define NUMBEROFALGORITHMS 2
char* namesOfMultiselectTimingFunctions[NUMBEROFALGORITHMS] = 
{"Sort and Choose Multiselect", "Bucket Multiselect"};

using namespace std;

namespace CompareMultiselect {

  /* This function compares bucketMultiselect with the other algorithms given in the
     defined range of kVals and array size.
  */
template<typename T>
void compareMultiselectAlgorithms(uint size, uint* kVals, uint numKs, uint numTests
, uint *algorithmsToTest, uint generateType, uint kGenerateType, char* fileNamecsv
, T* data = NULL) {

  // allocate space for operations
  T *h_vec, *h_vec_copy;
  float timeArray[NUMBEROFALGORITHMS][numTests];
  T * resultsArray[NUMBEROFALGORITHMS][numTests];
  float totalTimesPerAlgorithm[NUMBEROFALGORITHMS];
  uint winnerArray[numTests];
  uint timesWon[NUMBEROFALGORITHMS];
  uint i,j,m,x;
  int runOrder[NUMBEROFALGORITHMS];

  unsigned long long seed, seed2;
  results_t<T> *temp;
  ofstream fileCsv;
  timeval t1, t2;
 
  typedef results_t<T>* (*ptrToTimingFunction)(T*, uint, uint *, uint);
  typedef void (*ptrToGeneratingFunction)(T*, uint, hiprandGenerator_t);

  //these are the functions that can be called
  ptrToTimingFunction arrayOfTimingFunctions[NUMBEROFALGORITHMS] = 
    {&timeSortAndChooseMultiselect<T>,
     &timeBucketMultiselect<T>};
  
  ptrToGeneratingFunction *arrayOfGenerators;
  char** namesOfGeneratingFunctions;
  
  // this is the array of names of functions that generate problems of this type, 
  // ie float, double, or uint
  namesOfGeneratingFunctions = returnNamesOfGenerators<T>();
  arrayOfGenerators = (ptrToGeneratingFunction *) returnGenFunctions<T>();

  printf("Files will be written to %s\n", fileNamecsv);
  fileCsv.open(fileNamecsv, ios_base::app);
  
  //zero out the totals and times won
  bzero(totalTimesPerAlgorithm, NUMBEROFALGORITHMS * sizeof(uint));
  bzero(timesWon, NUMBEROFALGORITHMS * sizeof(uint));

  //allocate space for h_vec, and h_vec_copy
  h_vec = (T *) malloc(size * sizeof(T));
  h_vec_copy = (T *) malloc(size * sizeof(T));

  //create the random generators.
  hiprandGenerator_t generator;
  srand(unsigned(time(NULL)));

  printf("The distribution is: %s\n", namesOfGeneratingFunctions[generateType]);
  printf("The k distribution is: %s\n", namesOfKGenerators[kGenerateType]);

  /***********************************************/
  /*********** START RUNNING TESTS ************
  /***********************************************/

  for(i = 0; i < numTests; i++) {
    //hipDeviceReset();
    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
    
    for(m = 0; m < NUMBEROFALGORITHMS;m++)
      runOrder[m] = m;
    
    std::random_shuffle(runOrder, runOrder + NUMBEROFALGORITHMS);
    fileCsv << size << "," << numKs << "," << 
      namesOfGeneratingFunctions[generateType] << "," << 
      namesOfKGenerators[kGenerateType] << ",";

    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator,seed);
    printf("Running test %u of %u for size: %u and numK: %u\n", i + 1, 
           numTests, size, numKs);

    //generate the random vector using the specified distribution
    if(data == NULL) 
      arrayOfGenerators[generateType](h_vec, size, generator);
    else
      h_vec = data;

    //copy the vector to h_vec_copy, which will be used to restore it later
    memcpy(h_vec_copy, h_vec, size * sizeof(T));


    // if the kdistribution is random, we need to generate new a kList for each new random problem instance.
    if ( (kGenerateType != 1) && (i>0) ){
      gettimeofday(&t2, NULL);
      seed2 = t2.tv_usec * t2.tv_sec;
      hiprandGenerator_t generator2;
      srand(unsigned(time(NULL)));
      hiprandCreateGenerator(&generator2, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandSetPseudoRandomGeneratorSeed(generator2,seed2);

      arrayOfKDistributionGenerators[kGenerateType](kVals, numKs, size, generator2);
    }


    winnerArray[i] = 0;
    float currentWinningTime = INFINITY;
    //run the various timing functions
    for(x = 0; x < NUMBEROFALGORITHMS; x++){
      j = runOrder[x];
      if(algorithmsToTest[j]){

        //run timing function j
        printf("TESTING: %u\n", j);
        temp = arrayOfTimingFunctions[j](h_vec_copy, size, kVals, numKs);

        //record the time result
        timeArray[j][i] = temp->time;
        //record the value returned
        resultsArray[j][i] = temp->vals;
        //update the current "winner" if necessary
        if(timeArray[j][i] < currentWinningTime){
          currentWinningTime = temp->time;
          winnerArray[i] = j;
        }

        //perform clean up 
        free(temp);
        memcpy(h_vec_copy, h_vec, size * sizeof(T));
      }
    }

    hiprandDestroyGenerator(generator);
    for(x = 0; x < NUMBEROFALGORITHMS; x++)
      if(algorithmsToTest[x])
        fileCsv << namesOfMultiselectTimingFunctions[x] << "," << timeArray[x][i] << ",";

    // check for errors, and output information to recreate problem
    uint flag = 0;
    for(m = 1; m < NUMBEROFALGORITHMS;m++)
      if(algorithmsToTest[m])
        for (j = 0; j < numKs; j++) {
          if(resultsArray[m][i][j] != resultsArray[0][i][j]) {
            flag++;
            fileCsv << "\nERROR ON TEST " << i << " of " << numTests << " tests!!!!!\n";
            fileCsv << "vector size = " << size << "\nvector seed = " << seed << "\n";
            fileCsv << "numKs = " << numKs << "\n";
            fileCsv << "wrong k = " << kVals[j] << " kIndex = " << j << 
              " wrong result = " << resultsArray[m][i][j] << " correct result = " <<  
              resultsArray[0][i][j] << "\n";
            std::cout <<namesOfMultiselectTimingFunctions[m] <<
              " did not return the correct answer on test " << i + 1 << " at k[" << j << 
              "].  It got "<< resultsArray[m][i][j];
            std::cout << " instead of " << resultsArray[0][i][j] << ".\n" ;
            std::cout << "RESULT:\t";
            PrintFunctions::printBinary(resultsArray[m][i][j]);
            std::cout << "Right:\t";
            PrintFunctions::printBinary(resultsArray[0][i][j]);
          }
        }

    fileCsv << flag << "\n";
  }
  
  //calculate the total time each algorithm took
  for(i = 0; i < numTests; i++)
    for(j = 0; j < NUMBEROFALGORITHMS;j++)
      if(algorithmsToTest[j])
        totalTimesPerAlgorithm[j] += timeArray[j][i];

  //count the number of times each algorithm won. 
  for(i = 0; i < numTests;i++)
    timesWon[winnerArray[i]]++;

  printf("\n\n");

  //print out the average times
  for(i = 0; i < NUMBEROFALGORITHMS; i++)
    if(algorithmsToTest[i])
      printf("%-20s averaged: %f ms\n", namesOfMultiselectTimingFunctions[i], totalTimesPerAlgorithm[i] / numTests);

  for(i = 0; i < NUMBEROFALGORITHMS; i++)
    if(algorithmsToTest[i])
      printf("%s won %u times\n", namesOfMultiselectTimingFunctions[i], timesWon[i]);

  // free results
  for(i = 0; i < numTests; i++) 
    for(m = 0; m < NUMBEROFALGORITHMS; m++) 
      if(algorithmsToTest[m])
        free(resultsArray[m][i]);

  //free h_vec and h_vec_copy
  if(data == NULL) 
    free(h_vec);
  free(h_vec_copy);

  //close the file
  fileCsv.close();
}

  /* This function generates the array of kVals to work on and acts as a wrapper for 
     comparison.
   */
template<typename T>
void runTests (uint generateType, char* fileName, uint startPower, uint stopPower
, uint timesToTestEachK, uint kDistribution, uint startK, uint stopK, uint kJump) {
  uint algorithmsToRun[NUMBEROFALGORITHMS]= {1, 1};
  uint size;
  uint i;
  uint arrayOfKs[stopK+1];
  
  // double the array size to the next powers of 2
  for(size = (1 << startPower); size <= (1 << stopPower); size *= 2) {
    unsigned long long seed;
    timeval t1;
    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
    hiprandGenerator_t generator;
    srand(unsigned(time(NULL)));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator,seed);

    arrayOfKDistributionGenerators[kDistribution](arrayOfKs, stopK, size, generator);

    hiprandDestroyGenerator(generator);

    for(i = startK; i <= stopK; i+=kJump) {
      hipDeviceReset();

      printf("NOW ADDING ANOTHER K\n\n");
      compareMultiselectAlgorithms<T>(size, arrayOfKs, i, timesToTestEachK, 
                                      algorithmsToRun, generateType, kDistribution, fileName);
    }
  }
}
}

int main (int argc, char *argv[]) {

  using namespace CompareMultiselect;

  char *fileName, *hostName, *typeString;

  fileName = (char*) malloc(128 * sizeof(char));
  typeString = (char*) malloc(10 * sizeof(char));
  hostName = (char*) malloc(20 * sizeof(char));
  gethostname(hostName, 20);

  #ifdef GPUNUMBER
   hipSetDevice(GPUNUMBER);
  #endif


  time_t rawtime;
  struct tm * timeinfo;
  time ( &rawtime );
  timeinfo = localtime ( &rawtime );
  char * humanTime = asctime(timeinfo);
  humanTime[strlen(humanTime)-1] = '\0';

  uint testCount, type,distributionType,startPower,stopPower,kDistribution,startK
    ,stopK,jumpK;

  distributionType=0;
  startPower=26;
  stopPower=26;
  startK=100;
  jumpK=10;
  stopK=500;
  testCount=25;
  

  for(type=0; type<3; type++){
    for (kDistribution=0; kDistribution<5; kDistribution++){


      switch(type){
      case 0:
        typeString = "float";
        break;
      case 1:
        typeString = "double";
        break;
      case 2:
        typeString = "uint";
        break;
      default:
        break;
      } // end switch(type)

      snprintf(fileName, 128, 
               "%s %s k-dist:%s 2^%d to 2^%d (%d:%d:%d) %d-tests on %s at %s", 
               typeString, getDistributionOptions(type, distributionType), 
               getKDistributionOptions(kDistribution), startPower, stopPower, 
               startK, jumpK, stopK, testCount, hostName, humanTime);
      printf("File Name: %s \n", fileName);
 
      switch(type){
      case 0:
        runTests<float>(distributionType,fileName,startPower,stopPower,testCount,
                        kDistribution,startK,stopK,jumpK);
        break;
      case 1:
        runTests<double>(distributionType,fileName,startPower,stopPower,testCount,
                         kDistribution,startK,stopK,jumpK);
        break;
      case 2:
        runTests<uint>(distributionType,fileName,startPower,stopPower,testCount,
                       kDistribution,startK,stopK,jumpK);
        break;
      default:
        printf("You entered and invalid option, now exiting\n");
        break;
      } // end switch(type)

    } // end for(kDistribution)
  } // end for(type)


  free (fileName);
  return 0;
}

