#include "hip/hip_runtime.h"

/*

template<typename T>
struct minmax_pair
{
 T min_val;
 T max_val;
};


template<typename T>
struct MinMax_initializer
{
 __host__ __device__ __forceinline__ 
 minmax_pair<T> operator()(const T &a) const 
  {
    minmax_pair<T> result;
    result.min_val = a;
    result.max_val = a;
    return result;
  }
};


template<typename T>
struct MinMax_operator
{
 __host__ __device__ __forceinline__ 
 minmax_pair<T> operator()(const T &a, const minmax_pair<T> &b) const 
  {
    minmax_pair<T> result;
    result.min_val = hipcub::MIN(a, b.min_val);
    result.max_val = hipcub::MAX(a, b.max_val);
    return result;
  }
};
 

template<typename T>
void cubMinMax_alt(T* d_in, T* h_out, const int length)
{
  MinMax_initializer<T> init_op;
  minmax_pair<T> d_out=init_op(d_in[0]);
  //hipMalloc(&d_out, sizeof(T)*2);
  MinMax_operator<T> minmax_op;

  //initialize d_out
//  hipMemcpy(d_out, d_in, sizeof(T), hipMemcpyDeviceToDevice);
//  hipMemcpy(d_out+1, d_in, sizeof(T), hipMemcpyDeviceToDevice);
  

  // determine size of memory needed an allocate
  void *d_temp_storage = NULL;
  size_t temp_size = 0;
  hipcub::DeviceReduce::Reduce(d_temp_storage, temp_size, d_in, &d_out, length, minmax_op);
  hipMalloc(&d_temp_storage, temp_size);

  // find min and max
  hipcub::DeviceReduce::Reduce(d_temp_storage, temp_size, d_in, &d_out, length, minmax_op);

  // copy to host 
  hipMemcpy(h_out, d_out, 2*sizeof(T), hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_temp_storage);
  hipFree(d_out);
}


*/



template<typename T>
void cubMinMax_alt(T* d_in, T* h_out, const int length)
{
  T* d_out;
  hipMalloc(&d_out, sizeof(T)*2);

  void *d_temp_storage_min = NULL;
  size_t temp_size_min = 0;
  void *d_temp_storage_max = NULL;
  size_t temp_size_max = 0;

  hipStream_t *stream = (hipStream_t *)malloc(2*sizeof(hipStream_t));
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);

  // determine size of memory needed an allocate
  hipcub::DeviceReduce::Min(d_temp_storage_min, temp_size_min, d_in, d_out, length, stream[0]);
  hipMalloc(&d_temp_storage_min, 2*temp_size_min);
  hipcub::DeviceReduce::Min(d_temp_storage_min, temp_size_min, d_in, d_out, length, stream[0]);

  hipcub::DeviceReduce::Max(d_temp_storage_min+temp_size_min, temp_size_min, d_in, d_out+1, length, stream[1]);

/*
  hipcub::DeviceReduce::Max(d_temp_storage_max, temp_size_max, d_in, d_out+1, length, stream[1]);
  hipMalloc(&d_temp_storage_max, temp_size_max);
  hipcub::DeviceReduce::Max(d_temp_storage_max, temp_size_max, d_in, d_out+1, length, stream[1]);
*/
  // find min 
  //hipcub::DeviceReduce::Min(d_temp_storage, temp_size, d_in, d_out, length);

  // find max
  // hipcub::DeviceReduce::Max(d_temp_storage, temp_size, d_in, d_out+1, length);

  // copy to host 
  hipMemcpy(h_out, d_out, 2*sizeof(T), hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_temp_storage_min);
  hipFree(d_temp_storage_max);
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipFree(d_out);
}






